#include "hip/hip_runtime.h"
#include "memory_analysis.h"

__device__ int* constant_input;
__device__ int constant_thread_span;

__device__ int constant_num_threads;
__device__ int constant_num_elements;
__device__ int* constant_search_value;

__global__ void globalMemorySearch(int* input, int totalFound, int numElements,
                                   int numThreads) {
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadId < numThreads) {
    // Create a for loop that handles the fact that each thread needs to search for n values in input
    // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value

    input[threadId] += 1;
  }
}

__global__ void sharedMemorySearch(int* input, int totalFound, int numElements,
                                   int numThreads) {
  extern __shared__ int sharedInput[];
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadId < numThreads) {
    // Create a for loop that handles the fact that each thread needs to search for n values in input
    // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value
    input[threadId] += 1;
  }
}

__global__ void constantMemorySearch(int totalFound) {
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadId < constant_num_threads) {
    // Create a for loop that handles the fact that each thread needs to search for n values in input
    // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value
    constant_input[threadId] += 1;
  }
}

__global__ void registerMemorySearch(int* input, int totalFound,
                                     int numElements, int numThreads) {
  int threadId = blockDim.x * blockIdx.x + threadIdx.x;

  if (threadId < numThreads) {
    // Create a for loop that handles the fact that each thread needs to search for n values in input
    // Increment the input array at index i by 1, update in memory prior to testing if it is equal to constant_search_value

    input[threadId] += 1;
  }
}

// This will generate an array of size numElements of random integers from 0 to 255 in pageable host memory
__host__ int* allocatePageableRandomHostMemory(int numElements) {
  srand(time(0));
  size_t size = numElements * sizeof(int);

  // Allocate the host pinned memory input pointer B
  int* data;
  hipHostAlloc((void**)&data, size, hipHostMallocDefault);

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    data[i] = rand() % 255;
  }

  return data;
}

__host__ int* allocateDeviceMemory(int numElements) {
  size_t size = numElements * sizeof(int);

  int* d_input = NULL;
  hipError_t err = hipMalloc(&d_input, size);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to allocate device vector d_input (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return d_input;
}

__host__ void copyFromHostToDevice(std::string kernelType, int* input,
                                   int numElements, int numThreads,
                                   int* d_input) {
  size_t size = numElements * sizeof(int);

  if (!strcmp(kernelType.c_str(), "constant")) {
    // Copy input, numElements, threadSpan, and numThreads to constant memory

    hipMemcpyToSymbol((const char*)constant_input, input, size);
    hipMemcpyToSymbol((const char*)&constant_num_elements, &numElements,
                       sizeof(int));
    hipMemcpyToSymbol((const char*)&constant_num_threads, &numThreads,
                       sizeof(int));

    int threadSpan = numElements / numThreads;
    hipMemcpyToSymbol((const char*)&constant_thread_span, &threadSpan,
                       sizeof(int));
  } else {
    // Copy input to d_input
    hipError_t err = hipMemcpy(d_input, input, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
      fprintf(
          stderr,
          "Failed to copy array input from host to device (error code %s)!\n",
          hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }
  }
}

__host__ void executeKernel(int* d_input, int numElements, int threadsPerBlock,
                            std::string kernelType) {
  int totalFound = 0;
  // Launch the search CUDA Kernel
  if (!strcmp(kernelType.c_str(), "global")) {
    globalMemorySearch<<<1, threadsPerBlock>>>(d_input, totalFound, numElements,
                                               threadsPerBlock);
  } else if (!strcmp(kernelType.c_str(), "constant")) {
    constantMemorySearch<<<1, threadsPerBlock>>>(totalFound);
  } else if (!strcmp(kernelType.c_str(), "shared")) {
    unsigned int_array_size = numElements * sizeof(int);
    sharedMemorySearch<<<1, threadsPerBlock, int_array_size>>>(
        d_input, totalFound, numElements, threadsPerBlock);
  } else {
    registerMemorySearch<<<1, threadsPerBlock>>>(d_input, totalFound,
                                                 numElements, threadsPerBlock);
  }
  hipError_t err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch %s kernel (error code %s)!\n",
            kernelType.c_str(), hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipDeviceSynchronize();
}

// Free device global memory
__host__ void deallocateMemory(int* d_input) {

  hipError_t err = hipFree(d_input);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector d_input (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

// Reset the device and exit
__host__ void cleanUpDevice() {
  // hipDeviceReset causes the driver to clean up all state. While
  // not mandatory in normal operation, it is good practice.  It is also
  // needed to ensure correct operation when the application is being
  // profiled. Calling hipDeviceReset causes all profile data to be
  // flushed before the application exits
  hipError_t err = hipDeviceReset();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__host__ std::tuple<int, int, std::string> parseCommandLineArguments(
    int argc, char* argv[]) {
  int elementsPerThread = 2;
  int threadsPerBlock = 256;
  std::string currentPartId = "test";
  std::string kernelType = "global";

  for (int i = 1; i < argc; i++) {
    std::string option(argv[i]);
    i++;
    std::string value(argv[i]);
    if (option.compare("-t") == 0) {
      threadsPerBlock = atoi(value.c_str());
    } else if (option.compare("-m") == 0) {
      elementsPerThread = atoi(value.c_str());
    } else if (option.compare("-k") == 0) {
      kernelType = value;
    }
  }

  return {elementsPerThread, threadsPerBlock, kernelType};
}

__host__ int* setUpInput(int numElements) {
  srand(time(0));
  int* input;

  int searchValue = rand() % 255;
  hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constant_search_value), &searchValue,
                                       sizeof(int), 0, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy constant int d_v from host to device (error code "
            "%s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  input = allocatePageableRandomHostMemory(numElements);

  return input;
}

/*
 * Host main routine
 * -m elementsPerThread - the number of elements that a thread will search for a random value in
 * -t threadsPerBlock - the number of threads to schedule for concurrent processing
 * -k the kernel type - global, constant, shared, register
 */
int main(int argc, char* argv[]) {
  auto [elementsPerThread, threadsPerBlock, kernelType] =
      parseCommandLineArguments(argc, argv);

  int numElements = elementsPerThread * threadsPerBlock;

  int* input = setUpInput(numElements);
  int* d_input = allocateDeviceMemory(numElements);

  copyFromHostToDevice(kernelType, input, numElements, threadsPerBlock,
                       d_input);

  // Start time including kernel processing time
  hipEvent_t start, stop;
  float elapsedTime;

  hipEventCreate(&start);
  hipEventRecord(start, 0);

  executeKernel(d_input, numElements, threadsPerBlock, kernelType);

  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);

  deallocateMemory(d_input);
  cleanUpDevice();

  /*  
    std::ofstream outputfile;
    outputfile.open ("output.csv", std::ios_base::app);
    outputfile << "," << kernelType.c_str() << "," << threadsPerBlock << "," << elementsPerThread << "," << elapsedTime << "\n";
    outputfile.close();
*/
  std::cout << kernelType.c_str() << "," << threadsPerBlock << ","
            << elementsPerThread << "," << elapsedTime << "\n";

  return 0;
}
